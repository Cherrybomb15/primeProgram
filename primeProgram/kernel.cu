#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "math.h"
#include <iostream>
#include "Timer.h"

using namespace std;

#include <stdio.h>

__global__ void findPrimes(const int a, const int b, int* arr)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	const int c = i + a;

	bool go = true;

	if (c - 1 < b)
	{
		for (int k = 2; k < 7; k++)
		{
			if (c % k == 0)
			{
				arr[i] = 0;
				go = false;
			}
		}

		for (int j = 7; j < sqrtf(c) + 1 && go; j+=2)
		{
			if (c % j == 0)
			{
				arr[i] = 0;
				go = false;
			}
		}
	}

	if ((go && c - 1 < b) || c == 2 || c == 3 || c == 5)
	{
		arr[i] = c;
	}
}

int printPrimes(int* arr, int size)
{
	int k = 0;
	for (int i = 0; i < size; i++)
	{
		if (arr[i] != 0)
		{
			k++;
			//printf("%d ", arr[i]);
		}
	}

	return k;
}

int firstPrime(int* arr, int size)
{
	int i = 0;
	while (arr[i] == 0)
	{
		i++;
	}
	return arr[i];
}

int lastPrime(int* arr, int size)
{
	int i = size - 1;

	while (arr[i] == 0)
	{
		i--;
	}
	return arr[i];
}




int main()
{
	int a;
	int b;

	int comps;

	cout << "Lower bound: ";
	cin >> a;

	cout << "Upper bound: ";
	cin >> b;

	cout << "Threads per block: ";
	cin >> comps;

	Timer h;

	const int size = sizeof(int) * (b - a + 1);

	int* h_arr = (int*)malloc(size);

	int* d_arr;
	hipMalloc(&d_arr, size);

	hipMemcpy(d_arr, h_arr, size, hipMemcpyHostToDevice);

	findPrimes << <((b - a + 1) / comps) + 1, comps >> > (a, b, d_arr);
	//findPrimes << <gridSize, blockSize >> > (a, b, d_arr);

	hipMemcpy(h_arr, d_arr, size, hipMemcpyDeviceToHost);

	hipFree(d_arr);

	int lmoa = printPrimes(h_arr, b - a + 1);

	cout << "\nWhoa there are " << lmoa << " primes!!!" << "\n" <<
		"First prime: " << firstPrime(h_arr, b - a + 1) <<
		"\n Last prime: " << lastPrime(h_arr, b - a + 1);
	
	free(h_arr);

	cout << "\nTime passed:" << h.elapsed() <<
		"seconds. \n Blocks * threads: " <<
		(b - a + 1) / comps << " * " << comps;
}